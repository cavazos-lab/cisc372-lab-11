#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <sys/time.h>

const int N=1024;  /* multiple blocks used, change back from 512 to 1024 */

double rtclock()
{
   struct timezone Tzp;
   struct timeval Tp;

   int stat;
   stat = gettimeofday (&Tp, &Tzp);

   if (stat != 0) printf("Error return from gettimeofday: %d",stat);

   return(Tp.tv_sec + Tp.tv_usec*1.0e-6);
}

__global__ void matrixAdd(double *A, double *B, double *C, int N)
{
    int i, j;
    i = 0;
    j = blockIdx.x*blockDim.x + threadIdx.x; 

    if (j < N )
    {
      // The following statement is essentially: C[i][j] = A[i][j] + B[i][j]
      C[i*N+j] = A[i*N+j] + B[i*N+j];

      // Can also use the following
      //C[j*N+i] = A[j*N+i] + B[j*N+i];
    }
}

int main(int argc, char*argv[])
{
    double A[1][N];
    double *B;      /* Program Stack Size is limited, so only Array A could be statically allocated (on the stack) */
    double *C;      /* B and C arrays are to be dynamically allocated on the heap, which has much larger space */
	double *d_A, *d_B, *d_C;
    double * gpu_C;          /* stores the copy of d_C because CPU cannot access d_C directly
                                copy via hipMemcpy Device (i.e. d_C) -> Host (i.e. gpu_C) */

    int size = 1 * N * sizeof (double);
    B = (double *) malloc (size);

    C = (double *) malloc (size);

    int THREAD_DIMX,THREAD_DIMY,BLOCK_DIMX,BLOCK_DIMY;
    
    
    gpu_C= (double*) malloc (size);

	/* allocate space for device copies */
	hipMalloc( (void **) &d_A, size );
	hipMalloc( (void **) &d_B, size );
    hipMalloc( (void **) &d_C, size);

	for( int i = 0; i < 1; i++ )
        for( int j = 0; j < N; j++ )
	{
        A[i][j] = 1.0;
        B[i*N+j] = 2.0;
	}

	/* copy inputs to device */
	hipMemcpy( d_A, A, size, hipMemcpyHostToDevice );
	hipMemcpy( d_B, B, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */
    THREAD_DIMX = 32; 
    THREAD_DIMY = 1;

    BLOCK_DIMX = N/32;
    BLOCK_DIMY = 1; 

    dim3 dimGrid(BLOCK_DIMX,BLOCK_DIMY,1);
    dim3 dimBlock(THREAD_DIMX,THREAD_DIMY,1);
    
    double start_cpu = rtclock();

	matrixAdd<<< dimGrid, dimBlock>>>( d_A, d_B, d_C, N);

    hipDeviceSynchronize();


    double end_cpu = rtclock();
    printf("total time is %lf\n",(double)(end_cpu-start_cpu));  

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( gpu_C, d_C, size, hipMemcpyDeviceToHost );

    for (int i=0; i<1; i++)
        for (int j=0; j<N; j++)
        {
           C[i*N+j] = A[i][j] + B[i*N+j];
        if ( abs(C[i*N+j] - gpu_C[i*N+j]) > 1e-5 )
        {
            printf("CPU %f and GPU %f results do not match!\n", C[i*N+j], gpu_C[i*N+j]);
            exit(-1);
        } 
        }


	/* clean up */

	hipFree( d_A );
	hipFree( d_B );
	hipFree( d_C );
    free(gpu_C);
	
	return 0;
} /* end main */
